#include "hip/hip_runtime.h"
#include "particle.h"
#include "constant.h"
#include "linalg.h"

__host__ __device__ void Particle::updatePosition() {
    position += TIMESTEP * velocity;
}

__host__ __device__ void Particle::updateGradient() {
    velocity_gradient *= TIMESTEP;
    velocity_gradient.diagonal().array() += 1.0f;
    def_elastic = velocity_gradient * def_elastic;
}

__host__ __device__ void Particle::applyPlasticity() {
    Eigen::Matrix3f force_all(def_elastic * def_plastic);

    auto& u = svd_u;
    auto& s = svd_s;
    auto& v = svd_v;
    auto e = s.diagonal().array();

    linalg::svd3(def_elastic, u, s, v);

    // clip values
    e = e.min(CRIT_STRETCH).max(CRIT_COMPRESS);

#if ENABLE_IMPLICIT
    polar_r = u * v.transpose();
    polar_s = v;
    polar_s.diagonal().array() *= s.diagonal().array();
    polar_s = polar_s * v.transpose();
#endif

    Eigen::Matrix3f u_tmp(u), v_tmp(v);
    u_tmp.diagonal().array() *= e;
    v_tmp.diagonal().array() /= e;

    def_plastic = v_tmp * u.transpose() * force_all;
    def_elastic = u_tmp * v.transpose();
}

__host__ __device__ const Eigen::Matrix3f Particle::energyDerivative() {
    auto& u = svd_u;
    auto& s = svd_s;
    auto& v = svd_v;

    float harden = HARDENING * expf(1 - linalg::determinant(def_plastic)),
                   je = s.diagonal().prod();

    Eigen::Matrix3f tmp(2.0f * mu * (def_elastic - u * v.transpose()) * def_elastic.transpose());

    tmp.diagonal().array() += (lambda * je * (je - 1));

    return volume * harden * tmp;
}

#if ENABLE_IMPLICT
__host__ __device__ Eigen::Vector3f Particle::deltaForce(const Eigen::Vector3f& u, const Eigen::Vector3f& wg) {
    Matrix3f delta_elastic = TIMESTEP * (u * v.transpose()) * def_elastic;

    // TODO: wtf is the implicit math pdf???
}
#endif
